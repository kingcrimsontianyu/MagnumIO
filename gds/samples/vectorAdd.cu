/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>


#ifdef __cplusplus
extern "C" {
extern void vectorAdd(const float *A, const float *B, float *C,
                          int numElements);
}
#endif

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
extern "C"  __global__ void vectorAdd_kernel(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}

void vectorAdd(const float *d_A, const float *d_B, float *d_C,
                          int numElements) {
    // This is the new CUDA 4.0 API for Kernel Parameter Passing and Kernel Launch (simpler method)

    // Grid/Block configuration
    int threadsPerBlock = 256;
    int blocksPerGrid   = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    dim3 grid(blocksPerGrid, 1, 1);
    dim3 block(threadsPerBlock, 1, 1);

    // Launch the CUDA kernel
    vectorAdd_kernel<<< grid, block, 0 >>>(d_A, d_B, d_C, numElements);
    hipStreamSynchronize(0);
    return;
}
